#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <iostream>
#include <sstream>
#include <string>
#include <utility>
#include <fstream>

typedef struct {
  unsigned int term;
  unsigned int doc;
  unsigned int count;
} triple;

__global__ void
parallel_sort(triple *index_array, size_t array_size)
{
  // implementing simple odd-even transposition mergesort
  int sort_index = threadIdx.x * 2;
  if (sort_index >= array_size) {
    return;
  }

  for (int i = 0; i < array_size; i++) {
    auto current_index = sort_index + (i % 2);

    auto a = index_array[i];
    auto b = index_array[i + 1];
    if (current_index + 1 < array_size &&
        ((a.term == b.term && a.doc < b.doc)
         || a.term < b.term)) {
      auto temp = b;
      b = a;
      a = temp;
    }
     
    __syncthreads();
  }

  printf("%d %d %d; ", index_array[sort_index].term, index_array[sort_index].doc, index_array[sort_index].index);
}


int main(int argc, char **argv) {
  if(argc != 2) {
    std::cout << "Usage: " << argv[0] << std::endl;
    return;
  }

  std::ifstream triples_file(argv[1]);
  std::vector<triple> triples;

  if(!triples_file.is_open()) {
    std::cout << argv[1] << " is an invalid file" << std::endl;
    return;
  }
  
  std::string line;

  while(std::getline(triples_file, line)) {
    std::istringstream string_stream(line);
    std::string current_value;
    std::vector<unsigned int> current_triple;
    
    while(std::getline(string_stream, current_value, ',')) {
      current_triple.push_back(std::atoi(current_value.c_str()));
    }

    if(current_triple.size() != 3) {
      std::cout << "invalid length triple, ignoring" << std::endl;

      continue;
    }

    triples.emplace_back(triple({current_triple[0], current_triple[1], current_triple[2]}));
  }

  triple* d_triples;
  hipMalloc(&d_triples, triples.size() * sizeof(triple));
  hipMemcpy(d_triples, triples.data(), triples.size() * sizeof(triple), hipMemcpyHostToDevice);
  
  parallel_sort<<<1, triples.size() / 2>>>(triples.data(), triples.size());

  triple *out_triples = (triple *)malloc(sizeof(triple) * triples.size());
  hipMemcpy(out_triples, d_triples, triples.size() * sizeof(triple), hipMemcpyDeviceToHost);

  for (auto &i: triples) {
    std::cout << i.term << " " << i.doc << " " << i.count << std::endl;
  }

  return;
}